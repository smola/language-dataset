#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cm.h"
#include "zone_map.h"
#include "moderngpu/src/moderngpu/kernel_reduce.hxx"
#include "moderngpu/src/moderngpu/kernel_segreduce.hxx"


using namespace mgpu;
using namespace thrust::placeholders;

vector<void*> alloced_mem;

template<typename T>
struct distinct : public binary_function<T,T,T>
{
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return lhs != rhs;
    }
};



struct gpu_getyear
{
    const int_type *source;
    int_type *dest;

    gpu_getyear(const int_type *_source, int_type *_dest):
        source(_source), dest(_dest) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned long long int sec;
        uint quadricentennials, centennials, quadrennials, annuals/*1-ennial?*/;
        uint year, leap;
        uint yday;
        uint month, mday;
        const uint daysSinceJan1st[2][13]=
        {
            {0,31,59,90,120,151,181,212,243,273,304,334,365}, // 365 days, non-leap
            {0,31,60,91,121,152,182,213,244,274,305,335,366}  // 366 days, leap
        };
        unsigned long long int SecondsSinceEpoch = source[i]/1000;
        sec = SecondsSinceEpoch + 11644473600;

        //wday = (uint)((sec / 86400 + 1) % 7); // day of week
        quadricentennials = (uint)(sec / 12622780800ULL); // 400*365.2425*24*3600
        sec %= 12622780800ULL;

        centennials = (uint)(sec / 3155673600ULL); // 100*(365+24/100)*24*3600
        if (centennials > 3)
        {
            centennials = 3;
        }
        sec -= centennials * 3155673600ULL;

        quadrennials = (uint)(sec / 126230400); // 4*(365+1/4)*24*3600
        if (quadrennials > 24)
        {
            quadrennials = 24;
        }
        sec -= quadrennials * 126230400ULL;

        annuals = (uint)(sec / 31536000); // 365*24*3600
        if (annuals > 3)
        {
            annuals = 3;
        }
        sec -= annuals * 31536000ULL;

        year = 1601 + quadricentennials * 400 + centennials * 100 + quadrennials * 4 + annuals;
        leap = !(year % 4) && (year % 100 || !(year % 400));

        // Calculate the day of the year and the time
        yday = sec / 86400;
        sec %= 86400;
        //hour = sec / 3600;
        sec %= 3600;
        //min = sec / 60;
        sec %= 60;

        // Calculate the month
        for (mday = month = 1; month < 13; month++)
        {
            if (yday < daysSinceJan1st[leap][month])
            {
                mday += yday - daysSinceJan1st[leap][month - 1];
                break;
            }
        }
        dest[i] = year;
    }
};

struct gpu_getmonth
{
    const int_type *source;
    int_type *dest;

    gpu_getmonth(const int_type *_source, int_type *_dest):
        source(_source), dest(_dest) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned long long int sec;
        uint quadricentennials, centennials, quadrennials, annuals/*1-ennial?*/;
        uint year, leap;
        uint yday;
        uint month, mday;
        const uint daysSinceJan1st[2][13]=
        {
            {0,31,59,90,120,151,181,212,243,273,304,334,365}, // 365 days, non-leap
            {0,31,60,91,121,152,182,213,244,274,305,335,366}  // 366 days, leap
        };
        unsigned long long int SecondsSinceEpoch = source[i]/1000;
        sec = SecondsSinceEpoch + 11644473600;

        //wday = (uint)((sec / 86400 + 1) % 7); // day of week
        quadricentennials = (uint)(sec / 12622780800ULL); // 400*365.2425*24*3600
        sec %= 12622780800ULL;

        centennials = (uint)(sec / 3155673600ULL); // 100*(365+24/100)*24*3600
        if (centennials > 3)
        {
            centennials = 3;
        }
        sec -= centennials * 3155673600ULL;

        quadrennials = (uint)(sec / 126230400); // 4*(365+1/4)*24*3600
        if (quadrennials > 24)
        {
            quadrennials = 24;
        }
        sec -= quadrennials * 126230400ULL;

        annuals = (uint)(sec / 31536000); // 365*24*3600
        if (annuals > 3)
        {
            annuals = 3;
        }
        sec -= annuals * 31536000ULL;

        year = 1601 + quadricentennials * 400 + centennials * 100 + quadrennials * 4 + annuals;
        leap = !(year % 4) && (year % 100 || !(year % 400));

        // Calculate the day of the year and the time
        yday = sec / 86400;
        sec %= 86400;
        //hour = sec / 3600;
        sec %= 3600;
        //min = sec / 60;
        sec %= 60;

        // Calculate the month
        for (mday = month = 1; month < 13; month++)
        {
            if (yday < daysSinceJan1st[leap][month])
            {
                mday += yday - daysSinceJan1st[leap][month - 1];
                break;
            }
        }
        dest[i] = year*100+month;
    }
};


struct gpu_getday
{
    const int_type *source;
    int_type *dest;

    gpu_getday(const int_type *_source, int_type *_dest):
        source(_source), dest(_dest) {}
    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned long long int sec;
        uint quadricentennials, centennials, quadrennials, annuals/*1-ennial?*/;
        uint year, leap;
        uint yday;
        uint month, mday;
        const uint daysSinceJan1st[2][13]=
        {
            {0,31,59,90,120,151,181,212,243,273,304,334,365}, // 365 days, non-leap
            {0,31,60,91,121,152,182,213,244,274,305,335,366}  // 366 days, leap
        };
        unsigned long long int SecondsSinceEpoch = source[i]/1000;
        sec = SecondsSinceEpoch + 11644473600;

        //wday = (uint)((sec / 86400 + 1) % 7); // day of week
        quadricentennials = (uint)(sec / 12622780800ULL); // 400*365.2425*24*3600
        sec %= 12622780800ULL;

        centennials = (uint)(sec / 3155673600ULL); // 100*(365+24/100)*24*3600
        if (centennials > 3)
        {
            centennials = 3;
        }
        sec -= centennials * 3155673600ULL;

        quadrennials = (uint)(sec / 126230400); // 4*(365+1/4)*24*3600
        if (quadrennials > 24)
        {
            quadrennials = 24;
        }
        sec -= quadrennials * 126230400ULL;

        annuals = (uint)(sec / 31536000); // 365*24*3600
        if (annuals > 3)
        {
            annuals = 3;
        }
        sec -= annuals * 31536000ULL;

        year = 1601 + quadricentennials * 400 + centennials * 100 + quadrennials * 4 + annuals;
        leap = !(year % 4) && (year % 100 || !(year % 400));

        // Calculate the day of the year and the time
        yday = sec / 86400;
        sec %= 86400;
        //hour = sec / 3600;
        sec %= 3600;
        //min = sec / 60;
        sec %= 60;

        // Calculate the month
        for (mday = month = 1; month < 13; month++)
        {
            if (yday < daysSinceJan1st[leap][month])
            {
                mday += yday - daysSinceJan1st[leap][month - 1];
                break;
            }
        }
        dest[i] = year*10000+month*100+mday;
    }
};

void make_calc_columns(queue<string> op_type, queue<string> op_value, CudaSet* a, set<string>& order_field_names)
{
	string ss, s1_val;
    stack<string> exe_type, exe_value;
	string op_t, op_v;
	unsigned int bits;
	
	for(int i=0; !op_type.empty(); ++i, op_type.pop()) {
        ss = op_type.front();
		
		if (ss.compare("NAME") == 0) {
			if(!op_value.empty()) {
				exe_value.push(op_value.front());
				op_value.pop();	
			};
                }
		else if (ss.compare("CAST") == 0 || ss.compare("YEAR") == 0) {
			op_v = exe_value.top();
			exe_value.pop();
			op_t = ss;			
		}
		else if (ss.compare("emit sel_name") == 0) {
			if(!op_t.empty()) {
				
				if(cpy_bits.empty())
					bits = 0;
				else	
					bits = cpy_bits[op_v];			
			
				if(order_field_names.find(op_value.front()) == order_field_names.end()) {
					order_field_names.insert(op_value.front());
					order_field_names.erase(op_v);
				};	

				a->columnNames.push_back(op_value.front());
				a->cols[a->cols.size()+1] = op_value.front();
				a->type[op_value.front()] = 0;
				a->decimal[op_value.front()] = 0;
				a->decimal_zeroes[op_value.front()] = 0;
				
				
				a->h_columns_int[op_value.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >(a->mRecCount);
				a->d_columns_int[op_value.front()] = thrust::device_vector<int_type>(a->mRecCount);
				if (op_t.compare("CAST") == 0) {
					cpy_bits[op_value.front()] = bits;
					
					cpy_init_val[op_value.front()] = cpy_init_val[op_v]/100;
					if(bits == 8) {
						thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[op_v].data()));
						thrust::device_ptr<unsigned char> dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[op_value.front()].data()));
						thrust::transform(src, src + a->mRecCount, dest, _1/100);
					}
					else if(bits == 16) {
						thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[op_v].data()));
						thrust::device_ptr<unsigned short int> dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[op_value.front()].data()));
						thrust::transform(src, src + a->mRecCount, dest, _1/100);
					
					}
					else if(bits == 32) {
						thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[op_v].data()));
						thrust::device_ptr<unsigned int> dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[op_value.front()].data()));
						thrust::transform(src, src + a->mRecCount, dest, _1/100);					
					}
					else
						thrust::transform(a->d_columns_int[op_v].begin(), a->d_columns_int[op_v].begin() + a->mRecCount, a->d_columns_int[op_value.front()].begin(), _1/100);					
				}
				else {
					cpy_init_val[op_value.front()] = 0;
					cpy_bits[op_value.front()] = 0;
					if(bits == 8) {
						thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[op_v].data()));
						thrust::device_ptr<unsigned char> dest((unsigned char*)thrust::raw_pointer_cast(a->d_columns_int[op_value.front()].data()));
						thrust::copy(src, src + a->mRecCount, a->d_columns_int[op_value.front()].begin());
						thrust::transform(a->d_columns_int[op_value.front()].begin(), a->d_columns_int[op_value.front()].begin() + a->mRecCount, thrust::make_constant_iterator(cpy_init_val[op_v]), a->d_columns_int[op_value.front()].begin(), thrust::plus<int_type>());
						thrust::transform(a->d_columns_int[op_value.front()].begin(), a->d_columns_int[op_value.front()].begin() + a->mRecCount, a->d_columns_int[op_value.front()].begin(), _1/10000);
					}
					else if(bits == 16) {
						thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[op_v].data()));
						thrust::device_ptr<unsigned short int> dest((unsigned short int*)thrust::raw_pointer_cast(a->d_columns_int[op_value.front()].data()));
						thrust::transform(src, src + a->mRecCount, thrust::make_constant_iterator(10000), dest, thrust::divides<unsigned short int>());					
					}
					else if(bits == 32) {
						thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[op_v].data()));
						thrust::device_ptr<unsigned int> dest((unsigned int*)thrust::raw_pointer_cast(a->d_columns_int[op_value.front()].data()));
						thrust::transform(src, src + a->mRecCount, thrust::make_constant_iterator(10000), dest, thrust::divides<unsigned int>());				
					}
					else				
						thrust::transform(a->d_columns_int[op_v].begin(), a->d_columns_int[op_v].begin() + a->mRecCount, thrust::make_constant_iterator(10000), a->d_columns_int[op_value.front()].begin(), thrust::divides<int_type>());															
				};
				op_t.clear();
			};	
			op_value.pop();		
		}
		else  if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
			if(!exe_value.empty())
			    exe_value.pop();
			if(!exe_value.empty())
                            exe_value.pop();
		};	
		
	};
}

bool select(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums, queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a,
            CudaSet* b, vector<thrust::device_vector<int_type> >& distinct_tmp)
{

    stack<string> exe_type, exe_value;
    stack<int_type*> exe_vectors, exe_vectors1;
    stack<int_type> exe_nums, exe_nums1;
    string  s1, s2, s1_val, s2_val, grp_type;
    int_type n1, n2, res;
    unsigned int colCount = 0, dist_processed = 0;
    stack<int> col_type;
    stack<string> grp_type1, col_val, exe_value1;
    size_t res_size = 0;
    stack<float_type*> exe_vectors1_d;
    stack<unsigned int> exe_precision, exe_precision1;
    stack<bool> exe_ts;
    bool one_line = 0, ts, free_mem, free_mem1;

    //thrust::device_ptr<bool> d_di(thrust::raw_pointer_cast(a->grp.data()));

    if (a->grp_count && (a->mRecCount != 0))
        res_size = a->grp_count;

    std::clock_t start1 = std::clock();

    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();
        cout << ss << endl;

        if(ss.compare("emit sel_name") != 0) {
            grp_type = "NULL";

            if (ss.compare("COUNT") == 0  || ss.compare("SUM") == 0  || ss.compare("AVG") == 0 || ss.compare("MIN") == 0 || ss.compare("MAX") == 0 || ss.compare("DISTINCT") == 0 || ss.compare("YEAR") == 0 || ss.compare("MONTH") == 0 || ss.compare("DAY") == 0 || ss.compare("CAST") == 0) {

                if(!a->grp_count && ss.compare("YEAR") && ss.compare("MONTH") && ss.compare("DAY") && ss.compare("CAST")) {
                    one_line = 1;
                };

				if (ss.compare("CAST") == 0) {
					exe_type.push(ss);
					exe_value.push(op_value.front());
				}
				else if (ss.compare("YEAR") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    exe_type.pop();
                    thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(a->mRecCount);
                    if(a->ts_cols[s1_val]) {
                        thrust::counting_iterator<unsigned int> begin(0);
                        gpu_getyear ff((const int_type*)thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()),	thrust::raw_pointer_cast(res));
                        thrust::for_each(begin, begin + a->mRecCount, ff);
                        exe_precision.push(0);
						exe_vectors.push(thrust::raw_pointer_cast(res));
						exe_type.push("NAME");
						exe_value.push("");						
                    }
                    else {
						exe_type.push(ss);
						exe_value.push(op_value.front());						
                        exe_precision.push(a->decimal_zeroes[s1_val]);
                    };
                }
                else
                    if (ss.compare("MONTH") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();
                        exe_type.pop();
                        thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(a->mRecCount);
                        thrust::counting_iterator<unsigned int> begin(0);
                        gpu_getmonth ff((const int_type*)thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()),	thrust::raw_pointer_cast(res));
                        thrust::for_each(begin, begin + a->mRecCount, ff);
                        exe_precision.push(0);
                        exe_vectors.push(thrust::raw_pointer_cast(res));
                        exe_type.push("NAME");
                        exe_value.push("");
                    }
                    else
                        if (ss.compare("DAY") == 0) {
                            s1_val = exe_value.top();
                            exe_value.pop();
                            exe_type.pop();
                            thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(a->mRecCount);
                            thrust::counting_iterator<unsigned int> begin(0);
                            gpu_getday ff((const int_type*)thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()),	thrust::raw_pointer_cast(res));
                            thrust::for_each(begin, begin + a->mRecCount, ff);
                            exe_precision.push(0);
                            exe_vectors.push(thrust::raw_pointer_cast(res));
                            exe_type.push("NAME");
                            exe_value.push("");
                        }


                        else
                            if (ss.compare("DISTINCT") == 0) {
                                s1_val = exe_value.top();
                                exe_type.pop();
                                exe_value.pop();

                                if(a->type[s1_val] == 0) {

                                    thrust::copy(a->d_columns_int[s1_val].begin(), a->d_columns_int[s1_val].begin() + a->mRecCount,
                                                 distinct_tmp[dist_processed].begin());
                                    dist_processed++;
                                    thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(res_size);
                                    exe_vectors.push(thrust::raw_pointer_cast(res));
                                    exe_type.push("NAME");
                                    exe_value.push("");
                                }
                                else
                                    if(a->type[s1_val] == 2) {
                                        //will add a DISTINCT on strings if anyone needs it
                                        cout << "DISTINCT on strings is not supported yet" << endl;
                                        exit(0);
                                    }
                                    else {
                                        cout << "DISTINCT on float is not supported yet" << endl;
                                        exit(0);
                                    };
                            }

                            else
                                if (ss.compare("COUNT") == 0) {

                                    s1 = exe_type.top();
                                    //if(s1.compare("NAME") != 0) {  // non distinct

                                    grp_type = "COUNT";
                                    exe_type.pop();
                                    s1_val = exe_value.top();
                                    exe_value.pop();


                                    if (a->grp_count > 1) {
                                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                                        if(alloced_mem.empty()) {
                                            alloc_pool(a->maxRecs);
                                        };
                                        thrust::device_ptr<int_type> const_seq((int_type*)alloced_mem.back());
                                        thrust::fill(const_seq, const_seq+a->mRecCount, (int_type)1);
                                        segreduce(thrust::raw_pointer_cast(const_seq), a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), (int_type)0, context);
                                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                        exe_type.push("NAME");
                                    }
                                    else {
                                        thrust::device_ptr<int_type> dest  = thrust::device_malloc<int_type>(1);
                                        dest[0] = a->mRecCount;
                                        exe_vectors.push(thrust::raw_pointer_cast(dest));
                                        exe_type.push("NAME");
                                    };
                                    //    }
                                    //     else
                                    //         grp_type = "COUNTD";
                                    exe_precision.push(0);
                                    exe_value.push("");
                                }
                                else
                                    if (ss.compare("SUM") == 0) {

                                        /*if(op_case) {
                                        	cout << "found case " << endl;
                                        	op_case = 0;
                                        	while(!exe_type.empty())
                                        	{
                                        	cout << "CASE type " << exe_type.top() << endl;
                                        	exe_type.pop();
                                        	exit(0);
                                        	}

                                        };
                                        */

                                        grp_type = "SUM";
                                        s1 = exe_type.top();
                                        exe_type.pop();
                                        s1_val = exe_value.top();
                                        exe_value.pop();


                                        if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) == a->columnNames.end()) {
                                            int_type* s3 = exe_vectors.top();
                                            exe_vectors.pop();

                                            if (a->grp_count > 1) {
                                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                                                segreduce(s3, a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), (int_type)0, context);
                                                exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                            }
                                            else {
                                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(1);
                                                reduce(s3, a->mRecCount,  thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), context);
                                                exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                            };
                                            hipFree(s3);
                                        }
                                        else  {
                                            if (a->grp_count > 1) {
                                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                                                segreduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), (int_type)0, context);
                                                exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                            }
                                            else {
                                                thrust::device_ptr<int_type> dest;
                                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                                                reduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount,  thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), context);
                                                if (one_line) {
                                                    dest = thrust::device_malloc<int_type>(1);
                                                    dest[0] = count_diff[0];
                                                }
                                                else {
                                                    dest = thrust::device_malloc<int_type>(a->mRecCount);
                                                    int_type cc = count_diff[0];
                                                    thrust::sequence(dest, dest+(a->mRecCount), cc, (int_type)0);
                                                };
                                                exe_vectors.push(thrust::raw_pointer_cast(dest));

                                            };
                                            exe_precision.push(get_decimals(a, s1_val, exe_precision));
                                        }
                                        exe_type.push("NAME");
                                        exe_value.push("");
                                    }
                                    else
                                        if (ss.compare("MIN") == 0) {
                                            grp_type = "MIN";
                                            s1 = exe_type.top();
                                            exe_type.pop();
                                            s1_val = exe_value.top();
                                            exe_value.pop();
                                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);

                                            if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) != a->columnNames.end()) {
                                                if (a->grp_count > 1) {
                                                    segreduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), minimum_t<int_type>(), (int_type)0, context);
                                                }
                                                else {
                                                    reduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount,  thrust::raw_pointer_cast(count_diff), minimum_t<int_type>(), context);
                                                };
                                            }
                                            else {
                                                int_type* s3 = exe_vectors.top();
                                                exe_vectors.pop();
                                                if (a->grp_count > 1) {
                                                    segreduce(s3, a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), minimum_t<int_type>(), (int_type)0, context);
                                                }
                                                else {
                                                    reduce(s3, a->mRecCount,  thrust::raw_pointer_cast(count_diff), minimum_t<int_type>(), context);
                                                };
                                                hipFree(s3);
                                            };

                                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                            exe_type.push("NAME");
                                            exe_value.push("");
                                            exe_precision.push(get_decimals(a, s1_val, exe_precision));
                                        }
                                        else
                                            if (ss.compare("MAX") == 0) {
                                                grp_type = "MAX";
                                                s1 = exe_type.top();
                                                exe_type.pop();
                                                s1_val = exe_value.top();
                                                exe_value.pop();
                                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);

                                                if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) != a->columnNames.end()) {
                                                    if (a->grp_count > 1) {
                                                        segreduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), maximum_t<int_type>(), (int_type)0, context);
                                                    }
                                                    else {
                                                        reduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount,  thrust::raw_pointer_cast(count_diff), maximum_t<int_type>(), context);
                                                    };
                                                }
                                                else {
                                                    int_type* s3 = exe_vectors.top();
                                                    exe_vectors.pop();
                                                    if (a->grp_count > 1) {
                                                        segreduce(s3, a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), maximum_t<int_type>(), (int_type)0, context);
                                                    }
                                                    else {
                                                        reduce(s3, a->mRecCount,  thrust::raw_pointer_cast(count_diff), maximum_t<int_type>(), context);
                                                    };
                                                    hipFree(s3);
                                                };

                                                exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                                exe_type.push("NAME");
                                                exe_value.push("");
                                                exe_precision.push(get_decimals(a, s1_val, exe_precision));
                                            }

                                            else
                                                if (ss.compare("AVG") == 0) {

                                                    grp_type = "AVG";
                                                    s1 = exe_type.top();
                                                    exe_type.pop();
                                                    s1_val = exe_value.top();
                                                    exe_value.pop();
                                                    thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);

                                                    if(std::find(a->columnNames.begin(), a->columnNames.end(), s1_val) != a->columnNames.end()) {
                                                        if (a->grp_count > 1) {
                                                            segreduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), (int_type)0, context);
                                                        }
                                                        else {
                                                            reduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount,  thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), context);
                                                        };
                                                    }
                                                    else {
                                                        int_type* s3 = exe_vectors.top();
                                                        exe_vectors.pop();
                                                        if (a->grp_count > 1) {
                                                            segreduce(s3, a->mRecCount, (int*)thrust::raw_pointer_cast(a->grp.data()), a->grp.size(), thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), (int_type)0, context);
                                                        }
                                                        else {
                                                            reduce(s3, a->mRecCount,  thrust::raw_pointer_cast(count_diff), plus_t<int_type>(), context);
                                                        }
                                                        hipFree(s3);
                                                    };

                                                    exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                                    exe_type.push("NAME");
                                                    exe_value.push("");
                                                    exe_precision.push(get_decimals(a, s1_val, exe_precision));
                                                };
            };

            if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 ) {

                exe_type.push(ss);
                if (ss.compare("NUMBER") == 0) {
                    exe_nums.push(op_nums.front());
                    op_nums.pop();
                    exe_precision.push(op_nums_precision.front());
                    op_nums_precision.pop();
                }
                else
                    if (ss.compare("NAME") == 0) {
                        exe_value.push(op_value.front());
                        ts = a->ts_cols[op_value.front()];
                        op_value.pop();
                    }
            }
            else {
                if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                    // get 2 values from the stack
                    s1 = exe_type.top();
                    exe_type.pop();
                    s2 = exe_type.top();
                    exe_type.pop();

                    if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();
                        n2 = exe_nums.top();
                        exe_nums.pop();

                        auto p1 = exe_precision.top();
                        exe_precision.pop();
                        auto p2 = exe_precision.top();
                        exe_precision.pop();
                        auto pres = precision_func(p1, p2, ss);
                        exe_precision.push(pres);
                        if(p1)
                            n1 = n1*(unsigned int)pow(10,p1);
                        if(p2)
                            n2 = n2*(unsigned int)pow(10,p2);

                        if (ss.compare("ADD") == 0 )
                            res = n1+n2;
                        else
                            if (ss.compare("MUL") == 0 )
                                res = n1*n2;
                            else
                                if (ss.compare("DIV") == 0 )
                                    res = n1/n2;
                                else
                                    res = n1-n2;

                        thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                        thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);
                        exe_type.push("NAME");
                        exe_value.push("");
                        exe_vectors.push(thrust::raw_pointer_cast(p));
                    }

                    else
                        if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                            s1_val = exe_value.top();
                            exe_value.pop();
                            n1 = exe_nums.top();
                            exe_nums.pop();
                            auto p1 = exe_precision.top();
                            exe_precision.pop();
                            auto p2 = get_decimals(a, s1_val, exe_precision);
                            int_type* t = get_vec(a, s1_val, exe_vectors, free_mem);
                            auto pres = precision_func(p2, p1, ss);
                            exe_precision.push(pres);
                            exe_type.push("NAME");
                            exe_value.push("");
                            exe_vectors.push(a->op(t,n1,ss,1, p2, p1));
							if(free_mem)
								hipFree(t);								
                        }
                        else
                            if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                                n1 = exe_nums.top();
                                exe_nums.pop();
                                s1_val = exe_value.top();
                                exe_value.pop();
                                auto p1 = exe_precision.top();
                                exe_precision.pop();
                                auto p2 = get_decimals(a, s1_val, exe_precision);
                                int_type* t = get_vec(a, s1_val, exe_vectors, free_mem);
                                auto pres = precision_func(p2, p1, ss);
                                exe_precision.push(pres);
                                exe_type.push("NAME");
                                exe_value.push("");
                                exe_vectors.push(a->op(t,n1,ss,0, p2, p1));
								if(free_mem)
									hipFree(t);

                            }
                            else
                                if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                                    s1_val = exe_value.top();
                                    exe_value.pop();
                                    s2_val = exe_value.top();
                                    exe_value.pop();
                                    int_type* t1 = get_vec(a, s1_val, exe_vectors, free_mem);
                                    int_type* t = get_vec(a, s2_val, exe_vectors, free_mem1);
                                    auto p1 = get_decimals(a, s1_val, exe_precision);
                                    auto p2 = get_decimals(a, s2_val, exe_precision);
                                    auto pres = precision_func(p1, p2, ss);
                                    exe_precision.push(pres);
                                    exe_type.push("NAME");
                                    exe_value.push("");
                                    exe_vectors.push(a->op(t,t1,ss,0,p2,p1));
									if(free_mem)
										hipFree(t1);
									if(free_mem1)
										hipFree(t);

                                }
                }
            }

        }
        else {
            // here we need to save what is where

            col_val.push(op_value.front());
            op_value.pop();
            grp_type1.push(grp_type);

            if(!exe_nums.empty()) {  //number
                col_type.push(0);
                exe_nums1.push(exe_nums.top());
                exe_nums.pop();
                exe_precision1.push(exe_precision.top());
                exe_precision.pop();
            };
            if(!exe_value.empty() && exe_value.top() != "") {  //field name
                col_type.push(1);
                exe_precision1.push(a->decimal_zeroes[exe_value.top()]);
                exe_value1.push(exe_value.top());
                exe_ts.push(ts);
                exe_value.pop();
            };
            if(!exe_vectors.empty()) {  //vector int
                exe_vectors1.push(exe_vectors.top());
                exe_vectors.pop();
                col_type.push(2);
                exe_precision1.push(exe_precision.top());
                exe_precision.pop();
                exe_value.pop();
            };
            colCount++;
        };
    };



    for(unsigned int j=0; j < colCount; j++) {

        if ((grp_type1.top()).compare("COUNT") == 0 )
            b->grp_type[col_val.top()] = 0;
        else
            if ((grp_type1.top()).compare("AVG") == 0 )
                b->grp_type[col_val.top()] = 1;
            else
                if ((grp_type1.top()).compare("SUM") == 0 )
                    b->grp_type[col_val.top()] = 2;
                else
                    if ((grp_type1.top()).compare("NULL") == 0 )
                        b->grp_type[col_val.top()] = 3;
                    else
                        if ((grp_type1.top()).compare("MIN") == 0 )
                            b->grp_type[col_val.top()] = 4;
                        else
                            if ((grp_type1.top()).compare("MAX") == 0 )
                                b->grp_type[col_val.top()] = 5;
                            else
                                if ((grp_type1.top()).compare("COUNTD") == 0 ) {
                                    b->grp_type[col_val.top()] = 6;
                                };

        if(col_type.top() == 0) {
            // create a vector
            if (a->grp_count) {
                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                thrust::gather(a->grp.begin(), a->grp.end(), thrust::make_constant_iterator((int)exe_nums1.top()), count_diff);
                b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) , col_val.top(), res_size);
                thrust::device_free(count_diff);
            }
            else {
                thrust::device_ptr<int_type> s = thrust::device_malloc<int_type>(a->mRecCount);
                thrust::sequence(s, s+(a->mRecCount), (int)exe_nums1.top(), 0);
                b->addDeviceColumn(thrust::raw_pointer_cast(s), col_val.top(), a->mRecCount);
            }
            exe_nums1.pop();
            b->decimal_zeroes[col_val.top()] = exe_precision1.top();
            exe_precision1.pop();

        }
        else
            if(col_type.top() == 1) {

                //modify what we push there in case of a grouping
                if (a->grp_count) {
                    thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                    if(!exe_ts.top()) {
                        thrust::gather(a->grp.begin(), a->grp.end(), a->d_columns_int[exe_value1.top()].begin(), count_diff);
                    }
                    else {

                        thrust::device_vector<unsigned int> dd_tmp(res_size);
                        thrust::gather(a->grp.begin(), a->grp.end(), rcol_matches.begin(), count_diff);
                        thrust::gather(dd_tmp.begin(), dd_tmp.end(), rcol_dev.begin(), count_diff);

                    };
                    b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) ,  col_val.top(), res_size);
                    thrust::device_free(count_diff);
                }
                else
                    b->addDeviceColumn(thrust::raw_pointer_cast(a->d_columns_int[exe_value1.top()].data()) , col_val.top(), a->mRecCount);

                if(a->type[exe_value1.top()] == 0) {
                    b->decimal_zeroes[col_val.top()] = exe_precision1.top();
                    b->ts_cols[col_val.top()] = exe_ts.top();
                };

                if(a->type[exe_value1.top()] == 2 || (a->type[exe_value1.top()] == 0 && a->string_map.find(exe_value1.top()) != a->string_map.end())) {
                    b->string_map[col_val.top()] = a->string_map[exe_value1.top()];
                };
                exe_precision1.pop();
                exe_ts.pop();
                exe_value1.pop();
            }
            else
                if(col_type.top() == 2) {	    // int

                    if (a->grp_count)
                        b->addDeviceColumn(exe_vectors1.top() , col_val.top(), res_size);
                    else {
                        if(!one_line)
                            b->addDeviceColumn(exe_vectors1.top() , col_val.top(), a->mRecCount);
                        else
                            b->addDeviceColumn(exe_vectors1.top() , col_val.top(), 1);
                    };
                    hipFree(exe_vectors1.top());
                    exe_vectors1.pop();
                    b->decimal_zeroes[col_val.top()] = exe_precision1.top();
                    exe_precision1.pop();

                }
        col_type.pop();
        col_val.pop();
        grp_type1.pop();
    };

    if (!a->grp_count) {
        if(!one_line)
            b->mRecCount = a->mRecCount;
        else
            b->mRecCount = 1;
        return one_line;
    }
    else {
        b->mRecCount = res_size;
        return 0;
    };
}


