#include "hip/hip_runtime.h"
#include "luaT.h"
#include "THC.h"


#define MINUS_LOG_THRESHOLD -18.42
#define SPATIALLOGSOFTMAX_THREADS 128

// Parallelization across each feature point.
__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel
(float *output, float *input, int feature_size, int spatial_size, int data_size,
 float constant)
{
  int idx = (threadIdx.x + blockDim.x*blockIdx.x);
  idx = (idx/spatial_size)*feature_size + idx % spatial_size;

  if (idx < data_size) {
    int next_idx = idx + feature_size;
    float logsum = 0.0;
    float max = -2e38;
    // max
    for(int i = idx; i < next_idx; i += spatial_size) {
      if (max < input[i]) max = input[i];
    }

    // logsum
    for(int i = idx; i < next_idx; i += spatial_size) {
		if (!isnan(input[i])) {
		  logsum += __expf(input[i]-max);
	  	}
    }
		logsum += constant;
    logsum = __logf(logsum) + max;

    // logsoftmax
    for(int i = idx; i < next_idx; i += spatial_size){
      output[i] = input[i] - logsum;
    }
  }
}


__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int feature_size, int spatial_size, int data_size)
{
  int idx = (threadIdx.x + blockDim.x*blockIdx.x);
  idx = (idx/spatial_size)*feature_size + idx % spatial_size;

  if (idx < data_size) {
    int next_idx = idx + feature_size;
    float gradSum = 0.0;
    // Compute the sum of gradients
    for(int i = idx; i < next_idx; i += spatial_size){
      gradSum += gradOutput[i];
    }
    // Compute the new gradient
    for(int i = idx; i < next_idx; i += spatial_size){
      gradInput[i] = gradOutput[i] - __expf(output[i])*gradSum;
    }
  }
}

static int cunn_SpatialLogSoftMax_updateOutput(lua_State *L)
{
	THCState *state = getCutorchState(L);

	float constant = 0;
	if (luaT_getfieldcheckboolean(L, 1, "constant_present")) {
		constant = expf(luaT_getfieldchecknumber(L, 1, "constant"));
	}
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
//  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  int nframe = 0, dim = 0, height = 1, width = 1;

  if (input->nDimension == 1) {
    nframe = 1;
    dim = input->size[0];
  } else if(input->nDimension == 2) {
    nframe = input->size[0];
    dim = input->size[1];
  } else if (input->nDimension == 3) {
    nframe = 1;
    dim = input->size[0];
    width = input->size[1];
    height = input->size[2];
  } else if (input->nDimension == 4) {
    nframe = input->size[0];
    dim = input->size[1];
    width = input->size[2];
    height = input->size[3];
  } else {
    THArgCheck(0, 2, "4d input maximum expected");
  }

  // Get input and output
  input = THCudaTensor_newContiguous(state, input);
//  THCudaTensor_resizeAs(output, input);

  int spatial_size = width*height;
  int feature_size = dim*spatial_size;
  int data_size = feature_size*nframe;
  int nthreads = spatial_size*nframe;
  int nblocks = nthreads/SPATIALLOGSOFTMAX_THREADS;
  if (nthreads % SPATIALLOGSOFTMAX_THREADS != 0){
    nblocks = nblocks + 1;
  }

  dim3 blocks(nblocks,1,1);
  dim3 threads(SPATIALLOGSOFTMAX_THREADS,1,1);

  cunn_SpatialLogSoftMax_updateOutput_kernel<<<blocks,threads>>>
		(THCudaTensor_data(state, input), THCudaTensor_data(state, input),
		 feature_size, spatial_size, data_size, constant);

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, input);
  return 1;
}

static int cunn_SpatialLogSoftMax_updateGradInput(lua_State *L)
{
	THCState *state = getCutorchState(L);
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
//  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  int nframe = 0, dim = 0, height = 1, width = 1;
  
  if (output->nDimension == 1){
    nframe = 1;
    dim = output->size[0];
  } else if (output->nDimension == 2){
    nframe = output->size[0];
    dim = output->size[1];
  } else if (output->nDimension == 3){
    nframe = 1;
    dim = output->size[0];
    width = output->size[1];
    height = output->size[2];
  } else if (output->nDimension == 4){
    nframe = output->size[0];
    dim = output->size[1];
    width = output->size[2];
    height = output->size[3];
  } else {
    THError("4d output maximum expected");
  }

  //Get the data
  output = THCudaTensor_newContiguous(state, output);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
//  THCudaTensor_resizeAs(state, gradInput, output);

  int spatial_size = width*height;
  int feature_size = dim*spatial_size;
  int data_size = feature_size*nframe;
  int nthreads = spatial_size*nframe;
  int nblocks = nthreads/SPATIALLOGSOFTMAX_THREADS;
  if (nthreads % SPATIALLOGSOFTMAX_THREADS != 0){
    nblocks = nblocks + 1;
  }
  
  dim3 blocks(nblocks,1,1);
  dim3 threads(SPATIALLOGSOFTMAX_THREADS,1,1);

  cunn_SpatialLogSoftMax_updateGradInput_kernel<<<blocks,threads>>>(THCudaTensor_data(state, gradOutput), 
								    THCudaTensor_data(state, output), 
								    THCudaTensor_data(state, gradOutput),
								    feature_size, spatial_size, data_size);

  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, gradOutput);
  THCudaTensor_free(state, output);
  return 1;
}

static const struct luaL_Reg cunn_SpatialLogSoftMax__ [] = {
  {"SpatialLogSoftMax_updateOutput", cunn_SpatialLogSoftMax_updateOutput},
  {"SpatialLogSoftMax_updateGradInput", cunn_SpatialLogSoftMax_updateGradInput},
  {NULL, NULL}
};

void cunn_SpatialLogSoftMax_init(lua_State *L)
{
	luaL_openlib(L, "nn", cunn_SpatialLogSoftMax__, 0);
}
