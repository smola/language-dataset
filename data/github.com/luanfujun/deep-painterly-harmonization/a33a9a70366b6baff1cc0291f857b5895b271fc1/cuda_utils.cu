#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hip/hip_math_constants.h>
#include <math_functions.h>
#include <stdint.h>
#include <unistd.h>
#include <omp.h>
#include <getopt.h>
#include "hiprand/hiprand_kernel.h"

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__host__ __device__ int clamp(int x, int x_max, int x_min) {//assume x_max >= x_min
	if (x > x_max)
	{
		return x_max;
	}
	else if (x < x_min)
	{
		return x_min;
	}
	else
	{
		return x;
	}
}

__host__ __device__ int cuMax(int a, int b) {
	if (a > b) {
		return a;
	}
	else {
		return b;
	}
}
__host__ __device__ int cuMin(int a, int b) {
	if (a < b) {
		return a;
	}
	else {
		return b;
	}
}

__device__ float MycuRand(hiprandState &state) {//random number in cuda, between 0 and 1
	
	 return hiprand_uniform(&state);

}

__device__ void InitcuRand(hiprandState &state) {//random number in cuda, between 0 and 1
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(i, 0, 0, &state);

}

THCState* getCutorchState(lua_State* L)
{
	lua_getglobal(L, "cutorch");
	lua_getfield(L, -1, "getState");
	lua_call(L, 0, 1);
	THCState *state = (THCState*) lua_touserdata(L, -1);
	lua_pop(L, 2);
	return state;
}

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

THCudaTensor *new_tensor_like(THCState *state, THCudaTensor *x)
{
	THCudaTensor *y = THCudaTensor_new(state);
	THCudaTensor_resizeAs(state, y, x);
	return y;
}
 
__global__ void histogram_kernel(
	float *I, float *minI, float *maxI, float *mask, 
	int nbins, int c, int h, int w, float *hist
)
{
	int _id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w; 

	if (_id < c * size) {
		int id = _id % size, dc = _id / size;

		if (mask[id] < EPS)
			return ;

		float val  = I[_id];

		float _minI = minI[dc];
		float _maxI = maxI[dc];


		if (_minI == _maxI) {
			_minI -= 1;
			_maxI += 1;
		}

		if (_minI <= val && val <= _maxI) {
			int idx = MIN((val - _minI) / (_maxI - _minI) * nbins, nbins-1);
			int index = dc * nbins + idx;
			atomicAdd(&hist[index], 1.0f);
		}
		
	}

	return ;
}

int histogram(lua_State *L) {
	THCState *state     = getCutorchState(L);
	THCudaTensor *I     = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	int nbins           = luaL_checknumber(L, 2);
	THCudaTensor *minI  = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *maxI  = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *mask  = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");

	int c = THCudaTensor_size(state, I, 0);
	int h = THCudaTensor_size(state, I, 1);
	int w = THCudaTensor_size(state, I, 2);

	THCudaTensor *hist = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, hist, c, nbins);
	THCudaTensor_zero(state, hist);

	histogram_kernel<<<(c*h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, I),
		THCudaTensor_data(state, minI),
		THCudaTensor_data(state, maxI),
		THCudaTensor_data(state, mask),
		nbins, c, h, w,
		THCudaTensor_data(state, hist)
	);
	checkCudaError(L);	

	luaT_pushudata(L, hist, "torch.CudaTensor");
	return 1;
}

void histogram_cpu_kernel(
	float *I, float *minI, float *maxI, float *mask, 
	int nbins, int c, int h, int w, float *hist
)
{
	int size = h * w;
#pragma omp parallel for 
	for (int y = 0; y < h; y++)
		for (int x = 0; x < w; x++)
		{
			int id = y * w + x;
			if (mask[id] < EPS)
				continue;

			for (int dc = 0; dc < c; dc++)
			{
				float val   = I[dc * size + id];
				float _minI = minI[dc];
				float _maxI = maxI[dc];
				if (_minI == _maxI) {
					_minI -= 1;
					_maxI += 1;
				}
				if (_minI <= val && val <= _maxI) {
					int idx = MIN((val - _minI) / (_maxI - _minI) * nbins, nbins-1);
					int index = dc * nbins + idx;
					// atomicAdd(&hist[index], 1.0f);
					#pragma omp atomic 
					hist[index]++;
				}
			}
		}

	return ;
}

int histogram_cpu(lua_State *L) {
	// THCState *state     = getCutorchState(L);
	THFloatTensor *I    = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	int nbins           = luaL_checknumber(L, 2);
	THFloatTensor *minI  = (THFloatTensor*)luaT_checkudata(L, 3, "torch.FloatTensor");
	THFloatTensor *maxI  = (THFloatTensor*)luaT_checkudata(L, 4, "torch.FloatTensor");
	THFloatTensor *mask  = (THFloatTensor*)luaT_checkudata(L, 5, "torch.FloatTensor");

	int c = THFloatTensor_size(I, 0);
	int h = THFloatTensor_size(I, 1);
	int w = THFloatTensor_size(I, 2);

	THFloatTensor *hist = THFloatTensor_new();
	THFloatTensor_resize2d(hist, c, nbins);
	THFloatTensor_zero(hist);

	histogram_cpu_kernel(
		THFloatTensor_data(I),
		THFloatTensor_data(minI),
		THFloatTensor_data(maxI),
		THFloatTensor_data(mask),
		nbins, c, h, w,
		THFloatTensor_data(hist)
	);

	luaT_pushudata(L, hist, "torch.FloatTensor");
	return 1;
}

__global__ void hist_remap2_kernel(
	float *I, int nI, float *mI, float *histJ, float *cumJ, 
	float *_minJ, float *_maxJ, int nbins, 
	float *_sortI, int *_idxI, float *R, int c, int h, int w
)
{
	int _id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;

	if (_id < c * size) {
		// _id = dc * size + id
		int id = _id % size, dc = _id / size;

		float minJ  = _minJ[dc];
		float maxJ  = _maxJ[dc];
		float stepJ = (maxJ - minJ) / nbins;

		int idxI = _idxI[_id] - 1;
		if (mI[idxI] < EPS)
			return ;
		int offset = h * w - nI;
		 
		int cdf = id - offset;

		int s = 0;
		int e = nbins - 1;
		int m = (s + e) / 2;
		int binIdx = -1;

		while (s <= e) {
			// special handling for range boundary
			float cdf_e = m == nbins - 1 ? 
						  cumJ[dc * nbins + m] + 0.5f : 
						  cumJ[dc * nbins + m];
			float cdf_s = m == 0         ? 
						  -0.5f : 
						  cumJ[dc * nbins + m - 1];

			if (cdf >= cdf_e) {
				s = m + 1;
				m = (s + e) / 2;
			} else if (cdf < cdf_s) {
				e = m - 1;
				m = (s + e) / 2;
			} else {
				binIdx = m;    break;
			}
		}

		float hist  = histJ[dc * nbins + binIdx];
		float cdf_e = cumJ[dc * nbins + binIdx];
		float cdf_s = cdf_e - hist;
		float ratio = MIN(MAX((cdf - cdf_s) / (hist + 1e-8), 0.0f), 1.0f);
		float activation = minJ + (static_cast<float>(binIdx) + ratio) * stepJ;
		R[dc * size + idxI] = activation;
	}

	return ;
}

int hist_remap2(lua_State *L) {
	THCState *state       = getCutorchState(L);
	THCudaTensor *I       = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	int nI                = luaL_checknumber(L, 2);
	THCudaTensor *mI      = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *histJ   = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *cumJ    = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
	THCudaTensor *minJ    = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
	THCudaTensor *maxJ    = (THCudaTensor*)luaT_checkudata(L, 7, "torch.CudaTensor");
	int nbins             = luaL_checknumber(L, 8);
	THCudaTensor *sortI   = (THCudaTensor*)luaT_checkudata(L, 9, "torch.CudaTensor");
	THCudaIntTensor *idxI = (THCudaIntTensor*)luaT_checkudata(L, 10, "torch.CudaIntTensor");
	THCudaTensor *R       = (THCudaTensor*)luaT_checkudata(L, 11, "torch.CudaTensor");
	
	int c = THCudaTensor_size(state, I, 0);
	int h = THCudaTensor_size(state, I, 1);
	int w = THCudaTensor_size(state, I, 2);

	hist_remap2_kernel<<<(c*h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, I),
		nI,
		THCudaTensor_data(state, mI),
		THCudaTensor_data(state, histJ),
		THCudaTensor_data(state, cumJ),
		THCudaTensor_data(state, minJ),
		THCudaTensor_data(state, maxJ),
		nbins, 
		THCudaTensor_data(state, sortI),
		THCudaIntTensor_data(state, idxI),
		THCudaTensor_data(state, R),
		c, h, w
	);
	checkCudaError(L);	

	return 0;
}


__global__ void patchmatch_conv_kernel(
	float *input, float *target, float *conv, 
	int patch, int c1, int h1, int w1, int h2, int w2, 
	int *mask = NULL
)
{	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int size1 = h1 * w1, size2 = h2 * w2;
	int N = size1 * size2;

	if (id < N) {
		conv[id] = -1; 
		// id = id1 * size2 + id2
		int id1 = id / size2, id2 = id % size2;
		if (mask && mask[id1] == 0)
			return ;

		int x1 = id1 % w1, y1 = id1 / w1;
		int x2 = id2 % w2, y2 = id2 / w2;
		int kernel_radius  = (patch - 1) / 2;
		double conv_result = 0;
		// double sigma       = 0.5;
		// double sum_weight  = 0;
		// int cnt            = 0;
		for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
			for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
				int xx1 = x1 + dx, yy1 = y1 + dy;
				int xx2 = x2 + dx, yy2 = y2 + dy;	
				if (0 <= xx1 && xx1 < w1 && 0 <= yy1 && yy1 < h1 &&
					0 <= xx2 && xx2 < w2 && 0 <= yy2 && yy2 < h2) 
				{	
					int _id1 = yy1 * w1 + xx1, _id2 = yy2 * w2 + xx2;
					// float weight = exp(-(dx*dx + dy*dy) / (2*sigma*sigma));
					for (int c = 0; c < c1; c++) {
						float term1 = input[c * size1 + _id1];  
						float term2 = target[c * size2 + _id2];  
						conv_result += term1 * term2;
						// conv_result += (term1 - term2) * (term1 - term2) * weight;
					}	
					// cnt++;
					// sum_weight += weight;
				}
			}	
		}

		// conv[id] = conv_result / cnt;
		// conv[id] = conv_result / sum_weight;
		conv[id] = conv_result;
	}

	return ;
}

__global__ void patchmatch_argmax_kernel(
	float *conv, int *correspondence, int patch,
	int c1, int h1, int w1, int h2, int w2
) 
{	
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	int size1 = h1 * w1, size2 = h2 * w2;
	int kernel_radius = (patch - 1) / 2;
	if (id1 < size1) {
		float conv_max = -FLT_MAX;
		int y1 = id1 / w1, x1 = id1 % w1;

		for (int y2 = 0; y2 < h2; y2++) {
			for (int x2 = 0; x2 < w2; x2++) {
				int id2 = y2 * w2 + x2;
				int id = id1 * size2 + id2;
				float conv_result = conv[id];
				
				if (x2 < kernel_radius && !(x1 < kernel_radius))
					continue;
				if (x2 > w2 - 1 - kernel_radius && !(x1 > w1 - 1 - kernel_radius))
					continue;
				if (y2 < kernel_radius && !(y1 < kernel_radius))
					continue;
				if (y2 > h2 - 1 - kernel_radius && !(y1 > h1 - 1 - kernel_radius))
					continue;
				 
				if (conv_result > conv_max) {
					conv_max = conv_result;
					correspondence[id1 * 2 + 0] = x2;
					correspondence[id1 * 2 + 1] = y2;
				}
				// if (conv_result < conv_min) {
				// 	conv_min = conv_result;
				// 	correspondence[id1 * 2 + 0] = x2;
				// 	correspondence[id1 * 2 + 1] = y2;
				// }
			}
		}

	}

	return ;
}

int patchmatch(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *input  = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int           patch  = luaL_checknumber(L, 3);

	int c1 = THCudaTensor_size(state, input, 0);
	int h1 = THCudaTensor_size(state, input, 1);
	int w1 = THCudaTensor_size(state, input, 2);

	int c2 = THCudaTensor_size(state, target, 0);
	int h2 = THCudaTensor_size(state, target, 1);
	int w2 = THCudaTensor_size(state, target, 2);

	THCudaTensor *conv = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, conv, h1*w1, h2*w2);
	THCudaTensor_zero(state, conv);
	
	assert(c1 == c2);
	int N = h1*w1*h2*w2;
	patchmatch_conv_kernel<<<(N-1)/TB+1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, target),
		THCudaTensor_data(state, conv),
		patch, 
		c1,
		h1, w1,
		h2, w2
	);
	checkCudaError(L);

	THCudaIntTensor *correspondence = THCudaIntTensor_new(state);
	THCudaIntTensor_resize3d(state, correspondence, h1, w1, 2);
	THCudaIntTensor_zero(state, correspondence);

	patchmatch_argmax_kernel<<<(h1*w1-1)/TB+1, TB>>>(
		THCudaTensor_data(state, conv),
		THCudaIntTensor_data(state, correspondence),
		patch, 
		c1,
		h1, w1,
		h2, w2		
	);
	checkCudaError(L);	

	THCudaTensor_free(state, conv);

	luaT_pushudata(L, correspondence, "torch.CudaIntTensor");
	return 1;
}

int conv(lua_State *L) {
	THCState        *state  = getCutorchState(L);
	THCudaTensor    *input  = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor    *target = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int              patch  = luaL_checknumber(L, 3);
	THCudaIntTensor *mask   = (THCudaIntTensor*)luaT_checkudata(L, 4, "torch.CudaIntTensor");

	int c1 = THCudaTensor_size(state, input, 0);
	int h1 = THCudaTensor_size(state, input, 1);
	int w1 = THCudaTensor_size(state, input, 2);

	int c2 = THCudaTensor_size(state, target, 0);
	int h2 = THCudaTensor_size(state, target, 1);
	int w2 = THCudaTensor_size(state, target, 2);

	THCudaTensor *conv = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, conv, h1*w1, h2*w2);
	THCudaTensor_zero(state, conv);
	
	assert(c1 == c2);
	int N = h1*w1*h2*w2;
	patchmatch_conv_kernel<<<(N-1)/TB+1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, target),
		THCudaTensor_data(state, conv),
		patch, 
		c1,
		h1, w1,
		h2, w2,
		THCudaIntTensor_data(state, mask)
	);
	checkCudaError(L);

	luaT_pushudata(L, conv, "torch.CudaTensor");
	return 1;
}

__global__ void avg_vote_kernel(
	float *A, float *B, int *corrAB, 
	int patch, int c, int h, int w
)
{
	int _id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	int radius = patch / 2;
	if (_id < c * size) {
		// _id = dc * size + id
		int id = _id % size, dc = _id / size;
		int x1 = id % w, y1 = id / w;
		double sum = 0;
		int    cnt = 0;
		for (int dx = -radius; dx <= radius; dx++) {
			for (int dy = -radius; dy <= radius; dy++) {
				int new_x1 = x1 + dx, new_y1 = y1 + dy;
				
				if (new_x1 >= 0 && new_x1 < w && new_y1 >= 0 && new_y1 < h) {
					int new_id1 = new_y1 * w + new_x1;
					int x2 = corrAB[new_id1 * 2 + 0];
					int y2 = corrAB[new_id1 * 2 + 1];
					int new_x2 = x2 - dx, new_y2 = y2 - dy;

					if (new_x2 >= 0 && new_x2 < w && new_y2 >= 0 && new_y2 < h) {
						int new_id2 = new_y2 * w + new_x2;
						sum += A[dc * size + new_id2];
						cnt++;
					}
				}
			}
		}
		if (cnt != 0)
			B[dc * size + id] = sum / cnt;

	}
	return ;
}

int avg_vote(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaIntTensor *corrAB  = (THCudaIntTensor*)luaT_checkudata(L, 1, "torch.CudaIntTensor");
	THCudaTensor    *A       = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int              patch   = luaL_checknumber(L, 3);
 	THCudaTensor    *B       = new_tensor_like(state, A);
 	THCudaTensor_zero(state, B);

	int c = THCudaTensor_size(state, A, 0);
	int h = THCudaTensor_size(state, A, 1);
	int w = THCudaTensor_size(state, A, 2);

	// int h1 = THCudaIntTensor_size(state, corrAB, 0);
	// int w1 = THCudaIntTensor_size(state, corrAB, 1);
	// int c1 = THCudaIntTensor_size(state, corrAB, 2);

	avg_vote_kernel<<<(c*h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, A),
		THCudaTensor_data(state, B),
		THCudaIntTensor_data(state, corrAB),
		patch, c, h, w
	);
	checkCudaError(L);

	luaT_pushudata(L, B, "torch.CudaTensor");
	return 1;
}
 

__global__ void blend_kernel(
	float *A, float *BP, float *M, float *AP,
	float alpha, int c, int h, int w 
)
{
	int _id = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	if (_id < c * size) {
		// _id = dc * size + id
		int id = _id % size, dc = _id / size;
		// int x = id % w, y = id / w;
		float weight = M[id] < 0.05f ? 0.f : alpha;
		AP[dc * size + id] = 
			A[dc * size + id] * weight + 
			BP[dc * size + id] * (1.f - weight);
	}
	return ;
}

int blend(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor    *A       = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor    *BP      = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor    *M       = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	float            alpha   = luaL_checknumber(L, 4);

	THCudaTensor    *AP      = new_tensor_like(state, A);
	THCudaTensor_zero(state, AP);

	int c = THCudaTensor_size(state, A, 0);
	int h = THCudaTensor_size(state, A, 1);
	int w = THCudaTensor_size(state, A, 2);

	blend_kernel<<<(c*h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, A),
		THCudaTensor_data(state, BP),
		THCudaTensor_data(state, M),
		THCudaTensor_data(state, AP),
		alpha, c, h, w
	);
	checkCudaError(L);

	luaT_pushudata(L, AP, "torch.CudaTensor");
	return 1;
}

__global__ void patchmatch2_conv_kernel(
	float *A, float *B, float *AP, float *BP, float *conv, 
	int *prev_corrAB_upsampled, int patch, int s_rad, 
	int c, int h, int w
)
{	
	int h1 = h, h2 = h, w1 = w, w2 = w;
	int _id = blockIdx.x * blockDim.x + threadIdx.x;
	int size1 = h * w, size2 = h * w; 
	int s_size = 2 * s_rad + 1;
	int s_n = s_size * s_size; 
	if (_id < size1 * s_n) {
		conv[_id] = -1;

		int id1 = _id / s_n, s_idx = _id % s_n;
		int y1 = id1 / w1, x1 = id1 % w1;
		int dy2 = s_idx / s_size - s_rad, dx2 = s_idx % s_size - s_rad;

		int x2 = prev_corrAB_upsampled[2 * id1 + 0];
		int y2 = prev_corrAB_upsampled[2 * id1 + 1];

		int new_y2 = y2 + dy2;
		int new_x2 = x2 + dx2;
		if (!(new_x2 >= 0 && new_x2 < w2 && new_y2 >= 0 && new_y2 < h2)) {
			return ;
		}

		// Improve by local searching
		int kernel_radius = (patch - 1) / 2;
		float conv_result = 0;
		int cnt = 0;
		for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
			for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
				int xx1 = x1 + dx, yy1 = y1 + dy;
				int xx2 = new_x2 + dx, yy2 = new_y2 + dy;	
				if (0 <= xx1 && xx1 < w1 && 0 <= yy1 && yy1 < h1 &&
					0 <= xx2 && xx2 < w2 && 0 <= yy2 && yy2 < h2) 
				{
					int _id1 = yy1 * w1 + xx1, _id2 = yy2 * w2 + xx2;
					for (int dc = 0; dc < c; dc++) {
						float term1 = A[dc * size1 + _id1];  
						float term2 = B[dc * size2 + _id2];  
						conv_result += term1 * term2;

						term1 = AP[dc * size1 + _id1];  
						term2 = BP[dc * size2 + _id2];  
						conv_result += term1 * term2;
					}	
					cnt++;

				}
			}
		}

		conv[_id] = conv_result / cnt;
	}
	return ;
}

__global__ void patchmatch2_argmax_kernel(
	float *conv, int *prev_corrAB_upsampled, int *corrAB, int s_rad, 
	int c, int h, int w
)
{
	int h1 = h, h2 = h, w1 = w, w2 = w;
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	int size1 = h1 * w1;//, size2 = h2 * w2;
	int s_size = 2 * s_rad + 1;
	int s_n = s_size * s_size;

	if (id1 < size1) {
		float conv_max = -1;

		// int y1 = id1 / w1, x1 = id1 % w1;

		int x2 = prev_corrAB_upsampled[2 * id1 + 0];
		int y2 = prev_corrAB_upsampled[2 * id1 + 1];

		for (int dx2 = -s_rad; dx2 <= s_rad; dx2++) {
			for (int dy2 = -s_rad; dy2 <= s_rad; dy2++) {
				int new_y2 = y2 + dy2;
				int new_x2 = x2 + dx2;

				if (new_x2 >= 0 && new_x2 < w2 && new_y2 >= 0 && new_y2 < h2) {
					int s_idx = (dy2 + s_rad) * s_size + (dx2 + s_rad);
					int id = id1 * s_n + s_idx;
					float conv_result = conv[id];
					if (conv_result > conv_max) {
						conv_max = conv_result;
						corrAB[id1 * 2 + 0] = new_x2;
						corrAB[id1 * 2 + 1] = new_y2;
					}
				}
			}
		}
	}

	return ;
}


int patchmatch2(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *A  = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B  = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *AP = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *BP = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	int           patch  = luaL_checknumber(L, 5);
	int           s_rad  = luaL_checknumber(L, 6);
	THCudaIntTensor *prev_corrAB_upsampled = (THCudaIntTensor*)luaT_checkudata(L, 7, "torch.CudaIntTensor");

	int c = THCudaTensor_size(state, A, 0);
	int h = THCudaTensor_size(state, A, 1);
	int w = THCudaTensor_size(state, A, 2);

	THCudaIntTensor *corrAB = THCudaIntTensor_new(state);
	THCudaIntTensor_resize3d(state, corrAB, h, w, 2);
	THCudaIntTensor_zero(state, corrAB);

	THCudaTensor *conv = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, conv, h*w, (2*s_rad+1)*(2*s_rad+1));
	THCudaTensor_zero(state, conv);

	int N = h*w*(2*s_rad+1)*(2*s_rad+1);
	patchmatch2_conv_kernel<<<(N-1)/TB+1, TB>>>(
		THCudaTensor_data(state, A),
		THCudaTensor_data(state, B),
		THCudaTensor_data(state, AP),
		THCudaTensor_data(state, BP),
		THCudaTensor_data(state, conv),
		THCudaIntTensor_data(state, prev_corrAB_upsampled),
		patch, s_rad,
		c, h, w
	);
	checkCudaError(L);

	patchmatch2_argmax_kernel<<<(h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, conv),
		THCudaIntTensor_data(state, prev_corrAB_upsampled),
		THCudaIntTensor_data(state, corrAB),
		s_rad, c, h, w
	);
	checkCudaError(L);
 		
 	THCudaTensor_free(state, conv);

 	luaT_pushudata(L, corrAB, "torch.CudaIntTensor");
 	return 1;
}

__global__ void upsample_corr_kernel(
	int *curr_corr, int *next_corr, 
	int curr_h, int curr_w, int next_h, int next_w
)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < next_h * next_w) {
		int next_x = id % next_w, next_y = id / next_w;

		float w_ratio = (float)next_w / (float)curr_w;
		float h_ratio = (float)next_h / (float)curr_h;

		int curr_x = (next_x + 0.5) / w_ratio;
		int curr_y = (next_y + 0.5) / h_ratio;

		curr_x = MAX(MIN(curr_x, curr_w-1), 0);
		curr_y = MAX(MIN(curr_y, curr_h-1), 0);

		int curr_id = curr_y * curr_w + curr_x;
		
		int curr_x2 = curr_corr[2 * curr_id + 0];
		int curr_y2 = curr_corr[2 * curr_id + 1];

		int next_x2 = next_x + (curr_x2 - curr_x) * w_ratio + 0.5;
		int next_y2 = next_y + (curr_y2 - curr_y) * h_ratio + 0.5;

		next_x2 = MAX(MIN(next_x2, next_w-1), 0);
		next_y2 = MAX(MIN(next_y2, next_h-1), 0);

		next_corr[2 * id + 0] = next_x2;
		next_corr[2 * id + 1] = next_y2;
	}

	return ;
}

int upsample_corr(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaIntTensor *curr_corrAB = (THCudaIntTensor*)luaT_checkudata(L, 1, "torch.CudaIntTensor");
	int              next_h      = luaL_checknumber(L, 2);
	int              next_w      = luaL_checknumber(L, 3);
	THCudaIntTensor *next_corrAB = THCudaIntTensor_new(state);
	THCudaIntTensor_resize3d(state, next_corrAB, next_h, next_w, 2);
	THCudaIntTensor_zero(state, next_corrAB);

	int              curr_h      = THCudaIntTensor_size(state, curr_corrAB, 0);
	int              curr_w      = THCudaIntTensor_size(state, curr_corrAB, 1);

	upsample_corr_kernel<<<(next_h*next_w-1)/TB+1, TB>>>(
		THCudaIntTensor_data(state, curr_corrAB),
		THCudaIntTensor_data(state, next_corrAB),
		curr_h, curr_w, next_h, next_w
	);
	checkCudaError(L);

	luaT_pushudata(L, next_corrAB, "torch.CudaIntTensor");
	return 1;
}

__host__ __device__ float dist(float *A, float *B, float *AP, float *BP, 
	int x1, int y1, int x2, int y2, int c1, int h1, int w1, int h2, int w2, int patch_w) 
{
	int size1 = h1 * w1;
	int size2 = h2 * w2;
	float conv_result = 0;
	int cnt = 0;
	for (int dy = -patch_w / 2; dy <= patch_w / 2; dy++) {
		for (int dx = -patch_w / 2; dx <= patch_w / 2; dx++) {

			if (
				(y1 + dy) < h1 && (y1 + dy) >= 0 && (x1 + dx) < w1 && (x1 + dx) >= 0
				&&
				(y2 + dy) < h2 && (y2 + dy) >= 0 && (x2 + dx) < w2 && (x2 + dx) >= 0
				)
			{
				int _id1 = (y1 + dy) * w1 + (x1 + dx);
				int _id2 = (y2 + dy) * w2 + (x2 + dx);

				for (int c = 0; c < c1; c++) {
					float term1 = A[c * size1 + _id1];  
					float term2 = B[c * size2 + _id2];  
					conv_result += term1 * term2;

					term1 = AP[c * size1 + _id1];  
					term2 = BP[c * size2 + _id2];  
					conv_result += term1 * term2;
				}
				cnt++;
			}
		}
	}

	float d = conv_result / cnt;

	return d;
}

__device__ void improve_guess(float *A, float *B, float *AP, float *BP, 
	int x1, int y1, int x2, int y2, int c1, int h1, int w1, int h2, int w2, int patch_w, 
	int &xbest, int &ybest, float &dbest, float rr = 0.f)
{
	float d = dist(A, B, AP, BP, x1, y1, x2, y2, c1, h1, w1, h2, w2, patch_w);
	if (d > dbest + rr) { // note: normalized cross-correlation
		dbest = d;
		xbest = x2;
		ybest = y2;
	}	
}

#define ITERS 10
__global__ void PatchMatch_global_kernel(
	float *A, float *B, float *AP, float *BP, int *corr, int *prev_corr_upsampled,
	int patch, int rs_max, int c1, int h1, int w1, int h2, int w2
)
{
	int x1 = blockIdx.x*blockDim.x + threadIdx.x;
	int y1 = blockIdx.y*blockDim.y + threadIdx.y;
	int id1 = y1 * w1 + x1; 

	if (x1 < w1 && y1 < h1) {
		hiprandState state;
		InitcuRand(state);

		float dbest;
		int xbest, ybest;


		xbest = prev_corr_upsampled[2 * id1 + 0]; //static_cast<int>(MycuRand(state) * w2);
		ybest = prev_corr_upsampled[2 * id1 + 1]; //static_cast<int>(MycuRand(state) * h2);
		dbest = dist(A, B, AP, BP, x1, y1, xbest, ybest, c1, h1, w1, h2, w2, patch);
		corr[2 * id1 + 0] = xbest;
		corr[2 * id1 + 1] = ybest;

		for (int it = 0; it < ITERS; it++) 
		{	
			// Current best guess 
			xbest = corr[2 * id1 + 0];  
			ybest = corr[2 * id1 + 1];  
			dbest = dist(A, B, AP, BP, x1, y1, xbest, ybest, c1, h1, w1, h2, w2, patch);

			// Propagation
			for (int jump = 8; jump > 0; jump /= 2) {
				if (x1 - jump >= 0) { // left 
					int _id1 = y1 * w1 + x1 - jump;
					int xp = corr[2 * _id1 + 0] + jump;
					int yp = corr[2 * _id1 + 1];
					if (xp >= 0 && xp < w2 && yp >= 0 && yp < h2) {
						improve_guess(A, B, AP, BP, x1, y1, xp, yp, c1, h1, w1, h2, w2, patch, xbest, ybest, dbest);
						corr[2 * id1 + 0] = xbest;
						corr[2 * id1 + 1] = ybest;
					}
				}
				if (x1 + jump < w1) { // right 
					int _id1 = y1 * w1 + x1 + jump;
					int xp = corr[2 * _id1 + 0] - jump;
					int yp = corr[2 * _id1 + 1];
					if (xp >= 0 && xp < w2 && yp >= 0 && yp < h2) {
						improve_guess(A, B, AP, BP, x1, y1, xp, yp, c1, h1, w1, h2, w2, patch, xbest, ybest, dbest);
						corr[2 * id1 + 0] = xbest;
						corr[2 * id1 + 1] = ybest;
					}
				}
				if (y1 - jump >= 0) { // up 
					int _id1 = (y1 - jump) * w1 + x1;
					int xp = corr[2 * _id1 + 0];
					int yp = corr[2 * _id1 + 1] + jump;
					if (xp >= 0 && xp < w2 && yp >= 0 && yp < h2) {
						improve_guess(A, B, AP, BP, x1, y1, xp, yp, c1, h1, w1, h2, w2, patch, xbest, ybest, dbest);
						corr[2 * id1 + 0] = xbest;
						corr[2 * id1 + 1] = ybest;
					}
				}
				if (y1 + jump < h1) { // down 
					int _id1 = (y1 + jump) * w1 + x1;
					int xp = corr[2 * _id1 + 0];
					int yp = corr[2 * _id1 + 1] - jump;
					if (xp >= 0 && xp < w2 && yp >= 0 && yp < h2) {
						improve_guess(A, B, AP, BP, x1, y1, xp, yp, c1, h1, w1, h2, w2, patch, xbest, ybest, dbest);
						corr[2 * id1 + 0] = xbest;
						corr[2 * id1 + 1] = ybest;
					}
				}
			}

			// Random Search
			int rs = rs_max;
			if (rs > cuMax(h2, w2)) {
				rs = cuMax(h2, w2);
			}
			for (int mag = rs; mag >= 1; mag /= 2) {
				int xmin = cuMax(xbest - mag, 0),
				    xmax = cuMin(xbest + mag + 1, w2);
				int ymin = cuMax(ybest - mag, 0),
				    ymax = cuMin(ybest + mag + 1, h2);
				int xp   = xmin + (int)(MycuRand(state)*(xmax - xmin)) % (xmax - xmin);
				int yp   = ymin + (int)(MycuRand(state)*(ymax - ymin)) % (ymax - ymin);
				improve_guess(A, B, AP, BP, x1, y1, xp, yp, c1, h1, w1, h2, w2, patch, xbest, ybest, dbest, FLT_MIN);
				corr[2 * id1 + 0] = xbest;
				corr[2 * id1 + 1] = ybest;
			}

			__syncthreads();

		}
	}

	return ;
}

int PatchMatch(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *A  = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *B  = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *AP = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *BP = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	int           patch  = luaL_checknumber(L, 5);
	int           s_rad  = luaL_checknumber(L, 6);
	THCudaIntTensor *prev_corr_upsampled = (THCudaIntTensor*)luaT_checkudata(L, 7, "torch.CudaIntTensor");

	int c1 = THCudaTensor_size(state, A, 0);
	int h1 = THCudaTensor_size(state, A, 1);
	int w1 = THCudaTensor_size(state, A, 2);

	int c2 = THCudaTensor_size(state, BP, 0);
	int h2 = THCudaTensor_size(state, BP, 1);
	int w2 = THCudaTensor_size(state, BP, 2);

	THCudaIntTensor *corr = THCudaIntTensor_new(state);
	THCudaIntTensor_resize3d(state, corr, h1, w1, 2);
	THCudaIntTensor_zero(state, corr);

	// Generalized PatchMatch
	assert(c1 == c2);
	dim3 blocksPerGrid(w1 / 20 + 1, h1 / 20 + 1, 1);
	dim3 threadsPerBlock(20, 20, 1);

	PatchMatch_global_kernel<<<blocksPerGrid, threadsPerBlock>>>(
		THCudaTensor_data(state, A),
		THCudaTensor_data(state, B),
		THCudaTensor_data(state, AP),
		THCudaTensor_data(state, BP),
		THCudaIntTensor_data(state, corr),
		THCudaIntTensor_data(state, prev_corr_upsampled),
		patch, s_rad, 
		c1,
		h1, w1,
		h2, w2
	);
	checkCudaError(L);

	luaT_pushudata(L, corr, "torch.CudaIntTensor");
	return 1;
}

__global__ void Ring_kernel(
	float *A, float *BP, int *corrAB, float *M, 
	int ring, int c, int h, int w
)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	int ringSize  = 2*ring + 1;
	int ringPatch = ringSize * ringSize;
	if (id1 < size) {
		int y1 = id1 / w, x1 = id1 % w;
		int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
		// int id2 = y2 * w + x2;

		for (int dx = -ring; dx <= ring; dx++) 
			for (int dy = -ring; dy <= ring; dy++)
			{
				int pIdx = (dy + ring) * ringSize + (dx + ring);
				int _x2 = x2 + dx, _y2 = y2 + dy;
				if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h)
				{
					for (int dc = 0; dc < c; dc++) {
						// M[(dc * size + y1 * w + x1) * ringPatch + pIdx] = 
						M[(dc * size + y1 * w) * ringPatch + pIdx * w + x1] = 
							BP[dc * size + _y2 * w + _x2];
					}
				}
			}

	}

	return ;
}

int Ring(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *A         = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *BP        = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaIntTensor *corrAB = (THCudaIntTensor*)luaT_checkudata(L, 3, "torch.CudaIntTensor");
	int ring                = luaL_checknumber(L, 4);

	int c = THCudaTensor_size(state, A, 0);
	int h = THCudaTensor_size(state, A, 1);
	int w = THCudaTensor_size(state, A, 2);
 
	THCudaTensor *M = THCudaTensor_new(state);
	THCudaTensor_resize3d(state, M, c, h, w*(2*ring+1)*(2*ring+1));
 	THCudaTensor_zero(state, M); 

	Ring_kernel<<<(c*h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, A),
		THCudaTensor_data(state, BP),
		THCudaIntTensor_data(state, corrAB),
		THCudaTensor_data(state, M),
		ring, c, h, w
	);
	checkCudaError(L);

	luaT_pushudata(L, M, "torch.CudaTensor");
	return 1;
}

void Ring_cpu_kernel(
	float *A, float *BP, int *corrAB, float *M, 
	int ring, int c, int h, int w
)
{
	int ringSize  = 2*ring + 1;
	int ringPatch = ringSize * ringSize;
	int size      = h * w;

#pragma omp parallel for 
	for (int y1 = 0; y1 < h; y1++)
		for (int x1 = 0; x1 < w; x1++)
		{
			int id = y1 * w + x1;
			int x2 = corrAB[2*id + 0];
			int y2 = corrAB[2*id + 1];
			for (int dx = -ring; dx <= ring; dx++) 
				for (int dy = -ring; dy <= ring; dy++)
				{
					int pIdx = (dy + ring) * ringSize + (dx + ring);
					int _x2 = x2 + dx, _y2 = y2 + dy;
					if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h)
					{
						for (int dc = 0; dc < c; dc++) {
							M[(dc * size + y1 * w) * ringPatch + pIdx * w + x1] = 
								BP[dc * size + _y2 * w + _x2];
						}
					}
				}	

		}

	return ;
}

int Ring_cpu(lua_State *L) {
	THFloatTensor *A    = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	THFloatTensor *BP   = (THFloatTensor*)luaT_checkudata(L, 2, "torch.FloatTensor");
	THIntTensor *corrAB = (THIntTensor*)luaT_checkudata(L, 3, "torch.IntTensor");
	int ring            = luaL_checknumber(L, 4);

	int c = THFloatTensor_size(A, 0);
	int h = THFloatTensor_size(A, 1);
	int w = THFloatTensor_size(A, 2);
 
	THFloatTensor *M = THFloatTensor_new();
	THFloatTensor_resize3d(M, c, h, w*(2*ring+1)*(2*ring+1));
 	THFloatTensor_zero(M); 

	Ring_cpu_kernel(
		THFloatTensor_data(A),
		THFloatTensor_data(BP),
		THIntTensor_data(corrAB),
		THFloatTensor_data(M),
		ring, c, h, w
	);

	luaT_pushudata(L, M, "torch.FloatTensor");
	return 1;
}

__global__ void Ring2_kernel(
	float *A, float *BP, int *corrAB, int *mask, int *m, 
	int ring, int c, int h, int w
)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	if (id1 < size) {
		// int y1 = id1 / w, x1 = id1 % w;
		if (mask[id1] != 0) {

			int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
			for (int dx = -ring; dx <= ring; dx++) 
				for (int dy = -ring; dy <= ring; dy++)
				{
					int _x2 = x2 + dx, _y2 = y2 + dy;
					if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h)
					{
						m[_y2 * w + _x2] = 1;
					}
				}
		}
	}

	return ;
}

int Ring2(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *A         = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *BP        = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaIntTensor *corrAB = (THCudaIntTensor*)luaT_checkudata(L, 3, "torch.CudaIntTensor");
	int ring                = luaL_checknumber(L, 4);
	THCudaIntTensor *mask   = (THCudaIntTensor*)luaT_checkudata(L, 5, "torch.CudaIntTensor");

	int c = THCudaTensor_size(state, A, 0);
	int h = THCudaTensor_size(state, A, 1);
	int w = THCudaTensor_size(state, A, 2);
 
	THCudaIntTensor *m = THCudaIntTensor_new(state);
	THCudaIntTensor_resize2d(state, m, h, w);
	THCudaIntTensor_zero(state, m);

	Ring2_kernel<<<(h*w-1)/TB+1, TB>>>(
		THCudaTensor_data(state, A),
		THCudaTensor_data(state, BP),
		THCudaIntTensor_data(state, corrAB),
		THCudaIntTensor_data(state, mask),
		THCudaIntTensor_data(state, m),
		ring, c, h, w
	);
	checkCudaError(L);

	luaT_pushudata(L, m, "torch.CudaIntTensor");
	return 1;
}


__global__ void patchmatch_r_conv_kernel(
	float *input, float *target, float *conv, 
	int patch, int stride,  
	int c1, int h1, int w1, int h2, int w2
)
{	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int size1 = h1 * w1, size2 = h2 * w2;
	int N = size1 * size2;
	// id = id1 * size2 + id2

	if (id < N) {
		int id1 = id / size2, id2 = id % size2;

		int x1 = id1 % w1, y1 = id1 / w1;
		int x2 = id2 % w2, y2 = id2 / w2;

		int kernel_radius = (patch - 1) / 2;

		double conv_result = 0, norm_1 = 0, norm_2 = 0;
		for (int dy = -kernel_radius; dy <= kernel_radius; dy+=stride) {
			for (int dx = -kernel_radius; dx <= kernel_radius; dx+=stride) {
				int xx1 = x1 + dx, yy1 = y1 + dy;
				int xx2 = x2 + dx, yy2 = y2 + dy;	
				if (0 <= xx1 && xx1 < w1 && 0 <= yy1 && yy1 < h1 &&
					0 <= xx2 && xx2 < w2 && 0 <= yy2 && yy2 < h2) 
				{	
					int _id1 = yy1 * w1 + xx1, _id2 = yy2 * w2 + xx2;
					for (int c = 0; c < c1; c++) {
						float term1 = input[c * size1 + _id1];  
						float term2 = target[c * size2 + _id2];  
						conv_result += term1 * term2;
						norm_1      += term1 * term1;
						norm_2      += term2 * term2;
					}	

				}
			}	
		}

		norm_1 = sqrt(norm_1);
		norm_2 = sqrt(norm_2);

		conv[id] = conv_result / (norm_1 * norm_2 + 1e-9);		
	}

	return ;
}

__global__ void patchmatch_r_argmax_kernel(
	float *conv, float *target, float *match, int *correspondence,
	int c1, int h1, int w1, int h2, int w2
) 
{	
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	int size1 = h1 * w1, size2 = h2 * w2;
	
	if (id1 < size1) {
		//int x1 = id1 % w1, y1 = id1 / w1;
		double conv_max = -1e20;

		for (int y2 = 0; y2 < h2; y2++) {
			for (int x2 = 0; x2 < w2; x2++) {
				int id2 = y2 * w2 + x2;

				int id = id1 * size2 + id2;
				float conv_result = conv[id];

				if (conv_result > conv_max) {
					conv_max = conv_result;
					correspondence[id1 * 2 + 0] = x2;
					correspondence[id1 * 2 + 1] = y2;
					for (int c = 0; c < c1; c++) {
						match[c * size1 + id1] = target[c * size2 + id2];
					}
				}
			}
		}

	}

	return ;
}

int patchmatch_r(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *input  = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int           patch  = luaL_checknumber(L, 3);
	int 		  stride = luaL_checknumber(L, 4);

	int c1 = THCudaTensor_size(state, input, 0);
	int h1 = THCudaTensor_size(state, input, 1);
	int w1 = THCudaTensor_size(state, input, 2);

	int c2 = THCudaTensor_size(state, target, 0);
	int h2 = THCudaTensor_size(state, target, 1);
	int w2 = THCudaTensor_size(state, target, 2);

	THCudaTensor *conv = THCudaTensor_new(state);
	THCudaTensor_resize2d(state, conv, h1*w1, h2*w2);
	THCudaTensor_zero(state, conv);
	
	assert(c1 == c2);
	int N = h1*w1*h2*w2;
	patchmatch_r_conv_kernel<<<(N-1)/TB+1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, target),
		THCudaTensor_data(state, conv),
		patch, stride, 
		c1,
		h1, w1,
		h2, w2
	);
	checkCudaError(L);


	THCudaTensor *match = new_tensor_like(state, input);
	THCudaTensor_zero(state, match);

	THCudaIntTensor *correspondence = THCudaIntTensor_new(state);
	THCudaIntTensor_resize3d(state, correspondence, h1, w1, 2);
	THCudaIntTensor_zero(state, correspondence);

	
	patchmatch_r_argmax_kernel<<<(h1*w1-1)/TB+1, TB>>>(
		THCudaTensor_data(state, conv),
		THCudaTensor_data(state, target),
		THCudaTensor_data(state, match),
		THCudaIntTensor_data(state, correspondence),
		c1,
		h1, w1,
		h2, w2		
	);
	checkCudaError(L);	

	THCudaTensor_free(state, conv);

	luaT_pushudata(L, match, "torch.CudaTensor");
	luaT_pushudata(L, correspondence, "torch.CudaIntTensor");
	return 2;
}

__global__ void refineNNF_kernel(
	float *N_A, float *N_BP,
	int *init_corr, float *guide, 
	int *tmask, int *corr, 
	int patch, int c, int h, int w
)
{
	int id1 = blockIdx.x * blockDim.x + threadIdx.x;
	int size = h * w;
	int r = (patch - 1) / 2;
	if (id1 < size) {
		int x1 = id1 % w, y1 = id1 / w;
		int x2 = init_corr[2*id1 + 0];
		int y2 = init_corr[2*id1 + 1];

		corr[2*id1 + 0] = x2;
		corr[2*id1 + 1] = y2;

		if (tmask[id1] < EPS)
			return ;
		
		float best_d = FLT_MAX;
		int best_x2 = x2, best_y2 = y2;

		for (int dx = -r; dx <= r; dx++)
		for (int dy = -r; dy <= r; dy++)
		{
			int new_x1 = x1 + dx;
			int new_y1 = y1 + dy;
			int new_id1 = new_y1 * w + new_x1;
			if (new_x1 >= 0 && new_x1 < w && new_y1 >= 0 && new_y1 < h) {
				int new_x2 = init_corr[2*new_id1 + 0] - dx;
				int new_y2 = init_corr[2*new_id1 + 1] - dy;
				int new_id2 = new_y2 * w + new_x2;
				if (new_x2 >= r && new_x2 < w - r - 1 && new_y2 >= r && new_y2 < h - r - 1) {
					
					float dist = 0; 
					int cnt = 0;

					for (int _dx = -r; _dx <= r; _dx++)
					for (int _dy = -r; _dy <= r; _dy++)
					{
						int _new_x1 = x1 + _dx;
						int _new_y1 = y1 + _dy;
						int _new_id1 = _new_y1 * w + _new_x1;
						if (_new_x1 >= 0 && _new_x1 < w && _new_y1 >= 0 && _new_y1 < h) {
							int _new_x2 = init_corr[2*_new_id1 + 0] - _dx;
							int _new_y2 = init_corr[2*_new_id1 + 1] - _dy;
							int _new_id2 = _new_y2 * w + _new_x2;
							if (_new_x2 >= 0 && _new_x2 < w && _new_y2 >= 0 && _new_y2 < h) {
								float d = 0;
								for (int dc = 0; dc < 3; dc++) {
									float diff = guide[dc * size + new_id2] - guide[dc * size + _new_id2];
									d += diff * diff;
								}
								d = sqrt(d);
								dist += d;
								cnt++;
							}
						}
					}

					dist = dist / cnt;

					if (dist < best_d) {
						best_d = dist;
						best_x2 = new_x2;
						best_y2 = new_y2;
					}					


				}
			}
		}

		corr[2*id1 + 0] = best_x2;
		corr[2*id1 + 1] = best_y2;

	}
	return ;
}

int refineNNF(lua_State *L) {
	THCState        *state     = getCutorchState(L);
	THCudaTensor    *N_A       = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor    *N_BP      = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaIntTensor *init_corr = (THCudaIntTensor*)luaT_checkudata(L, 3, "torch.CudaIntTensor");
	THCudaTensor    *guide     = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaIntTensor *tmask     = (THCudaIntTensor*)luaT_checkudata(L, 5, "torch.CudaIntTensor");
	int              patch     = luaL_checknumber(L, 6);
	int              niter     = luaL_checknumber(L, 7);

	int c = THCudaTensor_size(state, N_BP, 0);
	int h = THCudaTensor_size(state, N_BP, 1);
	int w = THCudaTensor_size(state, N_BP, 2);
 
	THCudaIntTensor *corr = THCudaIntTensor_new(state);
	THCudaIntTensor_resize3d(state, corr, h, w, 2);
	THCudaIntTensor_zero(state, corr);

	for (int iter = 0; iter < niter; iter++) {
		printf("  iter=%d\n", iter);
		refineNNF_kernel<<<(h*w-1)/TB+1, TB>>>(
			THCudaTensor_data   (state, N_A),
			THCudaTensor_data   (state, N_BP),
			THCudaIntTensor_data(state, init_corr),
			THCudaTensor_data   (state, guide),
			THCudaIntTensor_data(state, tmask),
			THCudaIntTensor_data(state, corr),
			patch, c, h, w
		);
		checkCudaError(L);
		hipMemcpy(
			THCudaIntTensor_data(state, init_corr),
			THCudaIntTensor_data(state, corr),
			sizeof(int) * h * w * 2,
			hipMemcpyDeviceToDevice
		);
	}

	
		 
	luaT_pushudata(L, corr, "torch.CudaIntTensor");
	return 1;
}

static const struct luaL_Reg funcs[] = {
	{"histogram"    , histogram},     // compute histogram
	{"histogram_cpu", histogram_cpu}, // compute histogram on cpu
	{"hist_remap2"  , hist_remap2},   // histogram remapping
	{"patchmatch"   , patchmatch},    // brute force
	{"patchmatch_r" , patchmatch_r},  // raw
	{"conv"         , conv},          // brute force conv
	{"avg_vote"     , avg_vote},      // avg reconstruction features
	{"blend"        , blend},         // blend feature maps
	{"patchmatch2"  , patchmatch2},   // patch match locally
	{"upsample_corr", upsample_corr}, // upsample correspondence
	{"PatchMatch"   , PatchMatch},    // PatchMatch algorithm
	{"Ring"         , Ring},          // Spatial neural patch for more texture
	{"Ring_cpu"     , Ring_cpu},      // ... cpu
	{"Ring2"        , Ring2},         // draw on BP instead of A (no many-to-one since used once)
	{"refineNNF"    , refineNNF},     // NNF spatial consistency 
	{NULL, NULL}
};

extern "C" int luaopen_libcuda_utils(lua_State *L) {
	luaL_openlib(L, "cuda_utils", funcs, 0);
	return 1;
}