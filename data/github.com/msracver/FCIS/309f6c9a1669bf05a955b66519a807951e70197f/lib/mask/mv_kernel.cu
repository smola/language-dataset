#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fully Convolutional Instance-aware Semantic Segmentation
// Copyright (c) 2017 Microsoft
// Licensed under The MIT License [see LICENSE for details]
// Written by Haozhi Qi
// ------------------------------------------------------------------

#include "gpu_mv.hpp"
#include <iostream>

const int CAFFE_CUDA_NUM_THREADS = 512;

inline int CAFFE_GET_BLOCKS(const int N) {
  return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}


#define CUDA_POST_KERNEL_CHECK CUDA_CHECK(hipPeekAtLastError())

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

__device__ float bilinear_interpolate(const float* bottom_data,
                                      const int input_height, const int input_width,
                                      float inverse_y, float inverse_x) {

  // deal with cases that inverse elements are out of feature map boundary
  if (inverse_y <= 0) inverse_y = 0;
  if (inverse_x <= 0) inverse_x = 0;
  
  int h_low = (int) inverse_y;
  int w_low = (int) inverse_x;
  int h_high;
  int w_high;

  // handle boundary case
  if (h_low >= input_height - 1) {
    h_high = h_low = input_height - 1;
    inverse_y = (float) h_low;
  } else {
    h_high = h_low + 1;
  }

  if (w_low >= input_width - 1) {
    w_high = w_low = input_width - 1;
    inverse_x = (float) w_low;
  } else {
    w_high = w_low + 1;
  }

  float lh = inverse_y - h_low;
  float lw = inverse_x - w_low;
  float hh = 1 - lh, hw = 1 - lw;
  // corner point of interpolation
  float v1 = bottom_data[h_low * input_width + w_low];
  float v2 = bottom_data[h_low * input_width + w_high];
  float v3 = bottom_data[h_high * input_width + w_low];
  float v4 = bottom_data[h_high * input_width + w_high];
  // weight for each corner
  float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  // do bilinear interpolation
  float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

__global__ void mask_render(const int nthreads, const float* input_box, const float* input_mask, const int box_dim, const int mask_size,
                 const int image_height, const int image_width, float* target_buffer) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // target buffer's size if (n * h * w)
    int w = index % image_width;
    int h = (index / image_width) % image_height;
    int n = index / image_width / image_height;
    // get the n-th boxes
    const float* offset_box = input_box + n * box_dim;
    const float* offset_mask = input_mask + n * mask_size * mask_size;
    const float box_x1 = offset_box[0];
    const float box_y1 = offset_box[1];
    const float box_x2 = offset_box[2];
    const float box_y2 = offset_box[3];
    // check whether pixel is out of box bound
    if (w < box_x1 || w > box_x2 || h < box_y1 || h > box_y2) {
      target_buffer[index] = 0.0;
      continue;
    }
    const float box_width = box_x2 - box_x1 + 1.0;
    const float box_height = box_y2 - box_y1 + 1.0;
    const float ratio_w = (float) mask_size / box_width;
    const float ratio_h = (float) mask_size / box_height;
    const float inverse_x = ((float) w - box_x1 + 0.5) * ratio_w - 0.5;
    const float inverse_y = ((float) h - box_y1 + 0.5) * ratio_h - 0.5;

    target_buffer[index] = bilinear_interpolate(offset_mask, mask_size, mask_size, inverse_y, inverse_x);
  }
}

__global__ void mask_aggregate(const int nthreads, const float* render_mask, float* aggregate_mask, const int* candidate_inds, const int* candidate_starts, const float* candidate_weights, const int image_height, const int image_width, const float binary_thresh) {
  // render_mask: num_boxes * image_height * image_width
  // aggregate_mask: output_num * image_height * image_width
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % image_width;
    int h = (index / image_width) % image_height;
    int n = index / image_width / image_height;
    // get candidate_inds, candidate_start
    int candidate_start = (n == 0) ? 0 : candidate_starts[n-1];
    int candidate_end = candidate_starts[n];
    // output value will be summation of (mask * mask_weight)
    float val = 0.0;
    for (int i = candidate_start; i < candidate_end; ++i) {
      int input_mask_ind = candidate_inds[i];
      int offset_render_mask = (input_mask_ind * image_height + h) * image_width + w;
      const float mask_val = render_mask[offset_render_mask] >= binary_thresh ? 1.f : 0.f;
      val += (mask_val * candidate_weights[i]); 
    }
    aggregate_mask[index] = val;
  }
}

__global__ void reduce_mask_col(const int nthreads, const float* masks, int image_height, int image_width, const float binary_thresh, bool* output_buffer) {
  // nthreads will be output_num * image_width
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % image_width;
    int n = index / image_width;
    output_buffer[index] = false;
    for (int i = 0; i < image_height; ++i) {
      if (masks[(n * image_height + i) * image_width + w] >= binary_thresh) {
        output_buffer[index] = true;
        break;
      }
    }
  }
}

__global__ void reduce_mask_row(const int nthreads, const float* masks, int image_height, int image_width, const float binary_thresh, bool* output_buffer) {
  // nthreads will be output_num * image_width
  CUDA_KERNEL_LOOP(index, nthreads) {
    int h = index % image_height;
    int n = index / image_height;
    output_buffer[index] = false;
    for (int i = 0; i < image_width; ++i) {
      if (masks[(n * image_height + h) * image_width + i] >= binary_thresh) {
        output_buffer[index] = true;
        break;
      }
    }
  }
}

__global__ void reduce_bounding_x(const int nthreads, const bool* reduced_col, int* output_buffer, const int image_width) {
  // nthreads will be output_num * 2
  CUDA_KERNEL_LOOP(index, nthreads) {
    int x = index % 2;
    int n = index / 2;
    output_buffer[index] = image_width / 2;
    if (x == 0) {
      for (int i = 0; i < image_width; ++i) {
        if (reduced_col[n * image_width + i]) {
          output_buffer[index] = i;
          break;
        }
      }   
    } else {
      for (int i = image_width - 1; i >= 0; --i) {
        if (reduced_col[n * image_width + i]) {
          output_buffer[index] = i;
          break;
        }
      }
    }
  }
}

__global__ void reduce_bounding_y(const int nthreads, const bool* reduced_row, int* output_buffer, const int image_height) {
  // nthreads will be output_num * 2
  CUDA_KERNEL_LOOP(index, nthreads) {
    int x = index % 2;
    int n = index / 2;
    output_buffer[index] = image_height / 2;
    if (x == 0) {
      for (int i = 0; i < image_height; ++i) {
        if (reduced_row[n * image_height + i]) {
          output_buffer[index] = i;
          break;
        }
      }   
    } else {
      for (int i = image_height - 1; i >= 0; --i) {
        if (reduced_row[n * image_height + i]) {
          output_buffer[index] = i;
          break;
        }
      }
    }
  }
}


__global__ void mask_resize(const int nthreads, const float* original_mask, const int* bounding_x, const int* bounding_y, float* resized_mask, const int mask_size, const int image_height, const int image_width) {
  // output size should be result_num * mask_size * mask_size
  // original_mask should be result_num * image_height * image_width
  // bounding_x should be result_num * 2
  // bounding_y should be result_num * 2
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % mask_size;
    int h = (index / mask_size) % mask_size;
    int n = index / mask_size / mask_size;
    int bbox_x1 = bounding_x[n * 2];
    int bbox_x2 = bounding_x[n * 2 + 1];
    int bbox_y1 = bounding_y[n * 2];
    int bbox_y2 = bounding_y[n * 2 + 1];
    float bbox_width = bbox_x2 - bbox_x1 + 1.0;
    float bbox_height = bbox_y2 - bbox_y1 + 1.0;
    float ratio_w = bbox_width / static_cast<float>(mask_size);
    float ratio_h = bbox_height / static_cast<float>(mask_size);
    float inverse_x = bbox_x1 + static_cast<float>(w + 0.5) * ratio_w - 0.5;
    float inverse_y = bbox_y1 + static_cast<float>(h + 0.5) * ratio_h - 0.5;
    const float* offset_mask = original_mask + n * image_height * image_width;
    resized_mask[index] = bilinear_interpolate(offset_mask, image_height, image_width, inverse_y, inverse_x);
  }
}

void _mv(const float* all_boxes, const float* all_masks, const int all_boxes_num, const int* candidate_inds, const int* candidate_start, const float* candidate_weights, const int candidate_num, const float binary_thresh, const int image_height, const int image_width, const int box_dim, const int mask_size, const int result_num, float* finalize_output_mask, int* finalize_output_box, const int device_id) {

  // allocate device memory
  float* dev_boxes = NULL;
  float* dev_masks = NULL;
  int* dev_candidate_inds = NULL;
  float* dev_candidate_weights = NULL;
  int* dev_candidate_start = NULL;
  
  CUDA_CHECK(hipMalloc(&dev_boxes, all_boxes_num * box_dim * sizeof(float)));
  CUDA_CHECK(hipMemcpy(dev_boxes, all_boxes, all_boxes_num * box_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&dev_masks, all_boxes_num * mask_size * mask_size * sizeof(float)));
  CUDA_CHECK(hipMemcpy(dev_masks, all_masks, all_boxes_num * mask_size * mask_size * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&dev_candidate_inds, candidate_num * sizeof(int)));
  CUDA_CHECK(hipMemcpy(dev_candidate_inds, candidate_inds, candidate_num * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&dev_candidate_weights, candidate_num * sizeof(int)));
  CUDA_CHECK(hipMemcpy(dev_candidate_weights, candidate_weights, candidate_num * sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&dev_candidate_start, result_num * sizeof(int)));
  CUDA_CHECK(hipMemcpy(dev_candidate_start, candidate_start, result_num * sizeof(int),
                        hipMemcpyHostToDevice));

  // 1. Masks are of size mask_size x mask_size, to do aggregation
  //    first resize them to image scale (image_height x image_width)
  //    result n x image_height x image_width buffer
  const int render_mask_num = all_boxes_num * image_height * image_width;
  float* dev_render_mask = NULL;
  CUDA_CHECK(hipMalloc(&dev_render_mask, render_mask_num * sizeof(float)));
  
  mask_render<<<CAFFE_GET_BLOCKS(render_mask_num), CAFFE_CUDA_NUM_THREADS>>> (render_mask_num, dev_boxes, dev_masks, box_dim, mask_size, image_height, image_width, dev_render_mask);
  CUDA_POST_KERNEL_CHECK;

  // 2. After we get above buffer, we need to merge certain masks
  //    to get new masks according to candidate_weights and candidate_inds
  //    new_mask = \sum (old_mask * old_mask_weight)
  const int output_mask_num = result_num * image_height * image_width;
  float* dev_output_mask = NULL;
  CUDA_CHECK(hipMalloc(&dev_output_mask, output_mask_num * sizeof(float)));
  mask_aggregate<<<CAFFE_GET_BLOCKS(output_mask_num), CAFFE_CUDA_NUM_THREADS>>> (output_mask_num, dev_render_mask, dev_output_mask, dev_candidate_inds, dev_candidate_start, dev_candidate_weights, image_height, image_width, binary_thresh);

  CUDA_POST_KERNEL_CHECK;

  // 3. After we get new masks buffer (result_num * image_height * image_width)
  //    we then find the mask boundary, this is achieved by two reduction operation
  //    then the tight mask boundary can be obtained
  int reduced_col_num = result_num * image_width;
  bool* reduced_col_buffer = NULL;
  CUDA_CHECK(hipMalloc(&reduced_col_buffer, reduced_col_num * sizeof(bool)));
  reduce_mask_col<<<CAFFE_GET_BLOCKS(reduced_col_num), CAFFE_CUDA_NUM_THREADS>>> (reduced_col_num, dev_output_mask, image_height, image_width, binary_thresh, reduced_col_buffer);
  
  int reduced_bound_x_num = result_num * 2;
  int* reduced_bound_x = NULL;
  CUDA_CHECK(hipMalloc(&reduced_bound_x, reduced_bound_x_num * sizeof(int)));
  reduce_bounding_x<<<CAFFE_GET_BLOCKS(reduced_bound_x_num), CAFFE_CUDA_NUM_THREADS>>> (reduced_bound_x_num, reduced_col_buffer, reduced_bound_x, image_width);
  
  // find vertical boundary
  int reduced_row_num = result_num * image_height;
  bool* reduced_row_buffer = NULL;
  CUDA_CHECK(hipMalloc(&reduced_row_buffer, reduced_row_num * sizeof(bool)));
  reduce_mask_row<<<CAFFE_GET_BLOCKS(reduced_row_num), CAFFE_CUDA_NUM_THREADS>>> (reduced_row_num, dev_output_mask, image_height, image_width, binary_thresh, reduced_row_buffer);
  
  int reduced_bound_y_num = result_num * 2;
  int* reduced_bound_y = NULL;
  CUDA_CHECK(hipMalloc(&reduced_bound_y, reduced_bound_y_num * sizeof(int)));
  reduce_bounding_y<<<CAFFE_GET_BLOCKS(reduced_bound_y_num), CAFFE_CUDA_NUM_THREADS>>> (reduced_bound_y_num, reduced_row_buffer, reduced_bound_y, image_height);

  // 4. Once we get tight mask boundary, we could use it to resize masks back
  //    to mask_size x mask_size
  const int resized_mask_num = result_num * mask_size * mask_size;
  float* resized_mask = NULL;
  CUDA_CHECK(hipMalloc(&resized_mask, resized_mask_num * sizeof(float)));
  mask_resize<<<CAFFE_GET_BLOCKS(resized_mask_num), CAFFE_CUDA_NUM_THREADS>>> (resized_mask_num, dev_output_mask, reduced_bound_x, reduced_bound_y, resized_mask, mask_size, image_height, image_width);

  // copy back boxes to cpu
  int* cpu_bound_x = (int*) malloc(reduced_bound_x_num * sizeof(int));
  hipMemcpy(cpu_bound_x, reduced_bound_x, reduced_bound_x_num * sizeof(int), hipMemcpyDeviceToHost);
  int* cpu_bound_y = (int*) malloc(reduced_bound_y_num * sizeof(int));
  hipMemcpy(cpu_bound_y, reduced_bound_y, reduced_bound_y_num * sizeof(int), hipMemcpyDeviceToHost);
  int cnt = 0;
  for (int i = 0; i < result_num; i ++) {
    finalize_output_box[i*4] = cpu_bound_x[cnt];
    finalize_output_box[i*4+1] = cpu_bound_y[cnt];
    finalize_output_box[i*4+2] = cpu_bound_x[cnt+1];
    finalize_output_box[i*4+3] = cpu_bound_y[cnt+1];
    cnt += 2;
  }
  // copy back masks to cpu
  CUDA_CHECK(hipMemcpy(finalize_output_mask, resized_mask, resized_mask_num * sizeof(float), 
                        hipMemcpyDeviceToHost));
  
  // free gpu memories     
  CUDA_CHECK(hipFree(dev_boxes));
  CUDA_CHECK(hipFree(dev_masks));
  CUDA_CHECK(hipFree(dev_candidate_inds));
  CUDA_CHECK(hipFree(dev_candidate_start));
  CUDA_CHECK(hipFree(dev_candidate_weights));
  CUDA_CHECK(hipFree(dev_render_mask));
  CUDA_CHECK(hipFree(resized_mask));
  CUDA_CHECK(hipFree(dev_output_mask));
  CUDA_CHECK(hipFree(reduced_col_buffer));
  CUDA_CHECK(hipFree(reduced_bound_x));
  CUDA_CHECK(hipFree(reduced_row_buffer));
  CUDA_CHECK(hipFree(reduced_bound_y));
}
